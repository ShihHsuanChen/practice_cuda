#include "hip/hip_runtime.h"
#include<stdio.h>

/* Ref: https://en.wikipedia.org/wiki/Thread_block_(CUDA_programming)
The maximum x, y and z dimensions of a block are 1024, 1024 and 64, and it should be allocated
such that x × y × z ≤ 1024, which is the maximum number of threads per block.[3] Blocks can be
organized into one, two or three-dimensional grids of up to 231-1, 65,535 and 65,535 blocks in
the x, y and z dimensions respectively.[3] Unlike the maximum threads per block, there is not 
a blocks per grid limit distinct from the maximum grid dimensions.
*/
#define MAX_THREAD_X 1024;
#define MAX_THREAD_Y 1024;
#define MAX_THREAD_Z 64;
#define MAX_TOT_THREAD 1024;

// device code
// array (1D)
__global__ void kernal_add_const_1d(int *a, int c, int *o, int N) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < N) o[i] = a[i] + c;
}

__global__ void kernal_add_1d(int *a1, int *a2, int *o, int N) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < N) o[i] = a1[i] + a2[i];
}

__global__ void kernal_mul_1d(int* a1, int* a2, int* o, int N) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < N) o[i] = a1[i] * a2[i];
}

// matrix (2D)
__global__ void kernal_add_const_2d(int* a, int c, int *o, int N, int M) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    if (i < N && j < M) o[i*M+j] = a[i*M+j] + c;
}

__global__ void kernal_add_2d(int* a1, int *a2, int *o, int N, int M) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    if (i < N && j < M) o[i*M+j] = a1[i*M+j] + a2[i*M+j];
}

__global__ void kernal_mul_2d(int* a1, int *a2, int *o, int N, int M) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    if (i < N && j < M) o[i*M+j] = a1[i*M+j] * a2[i*M+j];
}

template<typename T>
__global__ void kernal_matmul_2d(T* a1, T *a2, T *o, int N, int H, int M) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    o[i*M+j] = 0;
    if (i < N && j < M) {
        for (int k=0;k<H;k++) o[i*M+j] += a1[i*M+k] * a2[k*N+j];
    }
}

// utils
void print_arr_1d(float *a, int N) {
    printf("[");
    for (int i=0;i<N-1;i++) printf("%f, ", a[i]);
    printf("%f]\n", a[N-1]);
}

void print_arr_2d(float *a, int N, int M) {
    printf("[");
    for (int i=0;i<N-1;i++) {
        print_arr_1d(&a[i*M], M);
        printf(" ");
    }
    printf("[");
    for (int j=0;j<M-1;j++) printf("%f, ", a[(N-1)*M+j]);
    printf("%f]]\n", a[N*M-1]);
}

float* create_device_arr_1d(int N) {
    float *d_arr;
    hipMalloc((void**)&d_arr, sizeof(float)*N);
    return d_arr;
}

float* create_device_arr_1d(int N, float* h_arr) {
    float *d_arr = create_device_arr_1d(N);
    hipMemcpy(d_arr, h_arr, sizeof(float)*N, hipMemcpyHostToDevice);
    return d_arr;
}

float* create_device_arr_2d(int N, int M) {
    return create_device_arr_1d(N*M);
}

float* create_device_arr_2d(int N, int M, float* h_arr) {
    return create_device_arr_1d(N*M, h_arr);
}

// main
int main() {
    int N = 3;
    int M = 3;
    // define array 1
    float h_arr1[N*M] = {
        0, 0, 0,
        1, 1, 1,
        2, 2, 2,
    };
    print_arr_2d(h_arr1, N, M); // TODO

    // define array 2
    float h_arr2[M*N] = {
        0, 1, 2,
        0, 1, 2,
        0, 1, 2,
    };
    print_arr_2d(h_arr2, M, N);

    // define output
    float h_arr_out[N*N];

    // define device pointers and copy host data to device
    float *d_arr1 = create_device_arr_2d(N, M, h_arr1);
    float *d_arr2 = create_device_arr_2d(M, N, h_arr2);
    float *d_arr_out = create_device_arr_2d(N, N); // NM * MN = NN

    // define kernal
    dim3 dim_grid(1);
    dim3 dim_block(M, N);

    // run kernal
    kernal_matmul_2d<<<dim_grid, dim_block>>>(d_arr1, d_arr2, d_arr_out, N, M, N);

    // copy from device to host
    hipMemcpy(h_arr_out, d_arr_out, sizeof(float)*N*N, hipMemcpyDeviceToHost);

    print_arr_2d(h_arr_out, N, N);
    return 0;
}
